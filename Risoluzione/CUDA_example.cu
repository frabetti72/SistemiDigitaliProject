//semplice programma CUDA che dovrebbe essere compilato con: nvcc CUDA_example.cu -o CUDA_example
//ed eseguito con ./CUDA_example


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(void) {
   printf("Hello, world! from GPU thread %d\n", threadIdx.x);
}

int main() {
   kernel<<<1, 10>>>(); // Lancia il kernel con 1 blocco e 10 thread

   hipDeviceSynchronize(); // Aspetta che tutti i thread del kernel finiscano

   return 0;
}